#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
// CUDA libraries.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// Include associated header file.
#include "../include/T5_NN.cuh"

__global__ void testGpu(float* x, float* A_row1, float* A_row2, float* A_row3, float* b)
{
    int row = blockIdx.x;
    int col = threadIdx.x;
    if (row == 0)
    {
        // printf("%d %d A_row1[%d] %f x[%d] %f %f b[row]:%f\n", row, col, col, A_row1[col], col, x[col], A_row1[col]*x[col], b[row]);
        b[row*44 + col] = A_row1[col] * x[col];
    }
    if (row == 1)
    {
        // printf("%d %d A_row2[%d] %f x[%d] %f %f b[row]:%f\n", row, col, col, A_row2[col], col, x[col], A_row2[col]*x[col], b[row]);
        b[row*44 + col] = A_row2[col] * x[col];
    }
    if (row == 2)
    {
        // printf("%d %d A_row3[%d] %f x[%d] %f %f b[row]:%f\n", row, col, col, A_row3[col], col, x[col], A_row3[col]*x[col], b[row]);
        b[row*44 + col] = A_row3[col] * x[col];
    }
    return;
}

void test()
{
    // To perform Ax = b
    // where A = 3 x 44 matrix, x = 44 x 1 vector
    // input values
    float cpu_host_x[44] = { 1 , 2 , 3 , 4 , 5  , 6  , 7  , 8  , 9  , 10 , 11 , 12 , 13 , 14 , 15 , 16 , 17 , 18 , 19 , 20 , 21 , 22 , 23 , 24 , 25 , 26 , 27 , 28 , 29 , 30 , 31 , 32 , 33 , 34 , 35 , 36 , 37 , 38 , 39 , 40 , 41 , 42 , 43 , 44 };
    float cpu_host_A_row1[44] = { 1 , 2 , 3 , 4 , 5  , 6  , 7  , 8  , 9  , 10 , 11 , 12 , 13 , 14 , 15 , 16 , 17 , 18 , 19 , 20 , 21 , 22 , 23 , 24 , 25 , 26 , 27 , 28 , 29 , 30 , 31 , 32 , 33 , 34 , 35 , 36 , 37 , 38 , 39 , 40 , 41 , 42 , 43 , 44 };
    float cpu_host_A_row2[44] = { 3 , 4 , 5 , 6 , 7  , 8  , 9  , 10 , 11 , 12 , 13 , 14 , 15 , 16 , 17 , 18 , 19 , 20 , 21 , 22 , 23 , 24 , 25 , 26 , 27 , 28 , 29 , 30 , 31 , 32 , 33 , 34 , 35 , 36 , 37 , 38 , 39 , 40 , 41 , 42 , 43 , 44 , 45 , 46 };
    float cpu_host_A_row3[44] = { 6 , 7 , 8 , 9 , 10 , 11 , 12 , 13 , 14 , 15 , 16 , 17 , 18 , 19 , 20 , 21 , 22 , 23 , 24 , 25 , 26 , 27 , 28 , 29 , 30 , 31 , 32 , 33 , 34 , 35 , 36 , 37 , 38 , 39 , 40 , 41 , 42 , 43 , 44 , 45 , 46 , 47 , 48 , 49 };
    float cpu_host_b[132]; // place to hold the result

    // Initialize x and allocate device memory for it
    float* gpu_device_x;
    float* gpu_device_A_row1;
    float* gpu_device_A_row2;
    float* gpu_device_A_row3;
    float* gpu_device_b; // memory space to hold results

    hipMalloc(&gpu_device_x, 44*sizeof(float));
    hipMalloc(&gpu_device_A_row1, 44*sizeof(float));
    hipMalloc(&gpu_device_A_row2, 44*sizeof(float));
    hipMalloc(&gpu_device_A_row3, 44*sizeof(float));
    hipMalloc(&gpu_device_b, 132*sizeof(float));

    // Asynchronous copy
    hipMemcpyAsync(gpu_device_x, cpu_host_x, 44*sizeof(float), hipMemcpyHostToDevice, /*stream=*/0);
    hipMemcpyAsync(gpu_device_A_row1, cpu_host_A_row1, 44*sizeof(float), hipMemcpyHostToDevice, /*stream=*/0);
    hipMemcpyAsync(gpu_device_A_row2, cpu_host_A_row2, 44*sizeof(float), hipMemcpyHostToDevice, /*stream=*/0);
    hipMemcpyAsync(gpu_device_A_row3, cpu_host_A_row3, 44*sizeof(float), hipMemcpyHostToDevice, /*stream=*/0);
    hipMemcpyAsync(gpu_device_b, cpu_host_b, 132*sizeof(float), hipMemcpyHostToDevice, /*stream=*/0);

    // Synchronizing
    hipStreamSynchronize(/*stream=*/0);

    dim3 blocksPerGrid(3, 1, 1);
    dim3 threadsPerBlock(44, 1, 1);

    // A                 times      x
    // <------------->              ^
    // <------------->              |
    // <------------->              V
    // three blocks
    // each with 44 threads

    testGpu<<<blocksPerGrid, threadsPerBlock>>>(gpu_device_x, gpu_device_A_row1, gpu_device_A_row2, gpu_device_A_row3, gpu_device_b);

    hipDeviceSynchronize();
    
    hipMemcpyAsync(cpu_host_b, gpu_device_b, 132*sizeof(float), hipMemcpyDeviceToHost, /*stream=*/0);

    // Synchronizing
    hipStreamSynchronize(/*stream=*/0);

    float result[3] = {0, 0, 0};
    for (int i = 0; i < 44; ++i)
    {
        result[0] += cpu_host_b[44 * 0 + i];
        result[1] += cpu_host_b[44 * 1 + i];
        result[2] += cpu_host_b[44 * 2 + i];
    }

    std::cout <<  " result[0]: " << result[0] <<  std::endl;
    std::cout <<  " result[1]: " << result[1] <<  std::endl;
    std::cout <<  " result[2]: " << result[2] <<  std::endl;

}
